#include <assert.h>
#include "diag_cusolver.cuh"
#include "hip/hip_runtime_api.h"

Diag_Cusolver_gvd::Diag_Cusolver_gvd(){
// step 1: create cusolver/cublas handle
    cusolverH = NULL;
    checkCudaErrors( hipsolverDnCreate(&cusolverH) );

    itype = HIPSOLVER_EIG_TYPE_1; // A*x = (lambda)*B*x
    jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    uplo = HIPBLAS_FILL_MODE_LOWER;

    d_A = NULL;
    d_B = NULL;
    d_work = NULL;

    d_A2 = NULL;
    d_B2 = NULL;
    d_work2 = NULL;

    d_W = NULL;
    devInfo = NULL;

    lwork = 0;
    info_gpu = 0;
    is_init = 0;
}

void Diag_Cusolver_gvd::finalize(){
// free resources and destroy
    if (d_A      ) {checkCudaErrors( hipFree(d_A) );  d_A  = NULL;}
    if (d_B      ) {checkCudaErrors( hipFree(d_B) );  d_B  = NULL;}
    if (d_A2     ) {checkCudaErrors( hipFree(d_A2) ); d_A2 = NULL;}
    if (d_B2     ) {checkCudaErrors( hipFree(d_B2) ); d_B2 = NULL;}
    if (d_W      ) {checkCudaErrors( hipFree(d_W) );  d_W  = NULL;}
    if (devInfo  ) {checkCudaErrors( hipFree(devInfo) );   devInfo = NULL;}
}

Diag_Cusolver_gvd::~Diag_Cusolver_gvd(){
    finalize();
    if (cusolverH) {checkCudaErrors( hipsolverDnDestroy(cusolverH) );    cusolverH = NULL;}
    checkCudaErrors( hipDeviceReset() );
}


void Diag_Cusolver_gvd::init_double(int N){
// step 2: Malloc A and B on device
    m = lda = N;
    checkCudaErrors( hipMalloc ((void**)&d_A, sizeof(double) * lda * m) );
    checkCudaErrors( hipMalloc ((void**)&d_B, sizeof(double) * lda * m) );
    checkCudaErrors( hipMalloc ((void**)&d_W, sizeof(double) * m) );
    checkCudaErrors( hipMalloc ((void**)&devInfo, sizeof(int)) );
}

void Diag_Cusolver_gvd::init_complex(int N){
// step 2: Malloc A and B on device
    m = lda = N;
    checkCudaErrors( hipMalloc ((void**)&d_A2, sizeof(hipDoubleComplex) * lda * m) );
    checkCudaErrors( hipMalloc ((void**)&d_B2, sizeof(hipDoubleComplex) * lda * m) ); 
    checkCudaErrors( hipMalloc ((void**)&d_W, sizeof(double) * m) );
    checkCudaErrors( hipMalloc ((void**)&devInfo, sizeof(int)) );
}
        
void Diag_Cusolver_gvd::Dngvd_double(int N, int M, double *A, double *B, double *W, double *V){

    // copy A, B to the GPU
        assert(N == M);
        if (M != m) {
            this->finalize();
            this->init_double(M);
        }
        checkCudaErrors( hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice) );
        checkCudaErrors( hipMemcpy(d_B, B, sizeof(double) * lda * m, hipMemcpyHostToDevice) );

    // Query working space of sygvd
    // The helper functions below can calculate the sizes needed for pre-allocated buffer.
    // The S and D data types are real valued single and double precision, respectively.
    // The C and Z data types are complex valued single and double precision, respectively.
        checkCudaErrors(hipsolverDnDsygvd_bufferSize(
            cusolverH,
            itype,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_B,
            lda,
            d_W,
            &lwork
        ));
        checkCudaErrors( hipMalloc((void**)&d_work, sizeof(double)*lwork) );

    // compute spectrum of (A,B)
        checkCudaErrors(hipsolverDnDsygvd(
            cusolverH,
            itype,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_B,
            lda,
            d_W,
            d_work,
            lwork,
            devInfo
        ));
        checkCudaErrors( hipDeviceSynchronize() );

    // copy (W, V) to the cpu root
        checkCudaErrors( hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost) );
        checkCudaErrors( hipMemcpy(V, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost) );
        checkCudaErrors( hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
        assert(0 == info_gpu);
    // free the buffer
        if (d_work ) checkCudaErrors( hipFree(d_work) );

}


void Diag_Cusolver_gvd::Dngvd_complex(int N, int M, std::complex<double> *A, std::complex<double> *B, double *W, std::complex<double> *V){
    
    // copy A, B to the GPU
        assert(N == M);
        if (M != m) {
            this->finalize();
            this->init_complex(M);
        }
        checkCudaErrors( hipMemcpy(d_A2, A, sizeof(hipDoubleComplex) * lda * m, hipMemcpyHostToDevice) );
        checkCudaErrors( hipMemcpy(d_B2, B, sizeof(hipDoubleComplex) * lda * m, hipMemcpyHostToDevice) );

    // Query working space of Zhegvd
    // The helper functions below can calculate the sizes needed for pre-allocated buffer.
    // The S and D data types are real valued single and double precision, respectively.
    // The C and Z data types are complex valued single and double precision, respectively.
        checkCudaErrors( 
            hipsolverDnZhegvd_bufferSize(
                cusolverH,
                itype,
                jobz,
                uplo,
                m,
                d_A2,
                lda,
                d_B2,
                lda,
                d_W,
                &lwork)
        );      
        checkCudaErrors( hipMalloc((void**)&d_work2, sizeof(hipDoubleComplex)*lwork) );

    // compute spectrum of (A,B)
        checkCudaErrors(
            hipsolverDnZhegvd(
                cusolverH,
                itype,
                jobz,
                uplo,
                m,
                d_A2,
                lda,
                d_B2,
                lda,
                d_W,
                d_work2,
                lwork,
                devInfo)
        );
        checkCudaErrors( hipDeviceSynchronize() );
        
    // copy (W, V) to the cpu root
        checkCudaErrors( hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost) );
        checkCudaErrors( hipMemcpy(V, d_A2, sizeof(std::complex<double>)*lda*m, hipMemcpyDeviceToHost) );
        checkCudaErrors( hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
        assert(0 == info_gpu);

    // free the buffer
        if (d_work2 ) checkCudaErrors( hipFree(d_work2) );
}
