#include "hip/hip_runtime.h"
#include "module_hsolver/kernels/dngvd_op.h"
#include "hip/hip_runtime_api.h"

#include <hipsolver.h>

#define cusolverErrcheck(res)                      \
    {                                              \
        cusolverAssert((res), __FILE__, __LINE__); \
    }

// cuSOLVER API errors
static const char* _cusolverGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }
    return "<unknown>";
}

inline void cusolverAssert(hipsolverStatus_t code, const char* file, int line, bool abort = true)
{
    if (code != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuSOLVER Assert: %s %s %d\n", _cusolverGetErrorEnum(code), file, line);
        if (abort)
            exit(code);
    }
}

namespace hsolver
{

static hipsolverHandle_t cusolver_H = nullptr;

void createCUSOLVERhandle()
{
    if (cusolver_H == nullptr)
    {
        cusolverErrcheck(hipsolverDnCreate(&cusolver_H));
    }
}

void destoryCUSOLVERhandle()
{
    if (cusolver_H != nullptr)
    {
        cusolverErrcheck(hipsolverDnDestroy(cusolver_H));
        cusolver_H = nullptr;
    }
}

static inline
void xhegvd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<float> * A, const int& lda,
        std::complex<float> * B, const int& ldb,
        float * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    float2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnChegvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const float2 *>(A), lda,
                                                 reinterpret_cast<const float2 *>(B), ldb, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(float2) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnChegvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<float2 *>(A), lda, reinterpret_cast<float2 *>(B), ldb, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

static inline
void xhegvd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<double> * A, const int& lda,
        std::complex<double> * B, const int& ldb,
        double * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const double2 *>(A), lda,
                                                 reinterpret_cast<const double2 *>(B), ldb, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(double2) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<double2 *>(A), lda, reinterpret_cast<double2 *>(B), ldb, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

static inline
void xheevd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<float> * A, const int& lda,
        float * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    float2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnCheevd_bufferSize(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const float2 *>(A), lda, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(float2) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnCheevd(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n, reinterpret_cast<float2 *>(A), lda, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

static inline
void xheevd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<double> * A, const int& lda,
        double * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnZheevd_bufferSize(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const double2 *>(A), lda, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(double2) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZheevd(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<double2 *>(A), lda, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

template <typename FPTYPE>
struct dngvd_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(
            const psi::DEVICE_GPU *d,
            const int nstart,
            const int ldh,
            const std::complex<FPTYPE> *A, // hcc
            const std::complex<FPTYPE> *B, // scc
            FPTYPE *W, // eigenvalue
            std::complex<FPTYPE> *V)
    {
        assert(nstart == ldh);
        // A to V
        checkCudaErrors(hipMemcpy(V, A, sizeof(std::complex<FPTYPE>) * ldh * nstart, hipMemcpyDeviceToDevice));
        xhegvd_wrapper(HIPBLAS_FILL_MODE_UPPER, nstart, V, ldh,
                       (std::complex<FPTYPE> *)B, ldh, W);
    }
};

template <typename FPTYPE>
struct dnevx_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(
            const psi::DEVICE_GPU *d,
            const int nstart,
            const int ldh,
            const std::complex<FPTYPE> *A, // hcc
            const int m,
            FPTYPE *W, // eigenvalue
            std::complex<FPTYPE> *V)
    {
        assert(nstart <= ldh);
        // A to V
        checkCudaErrors(hipMemcpy(V, A, sizeof(std::complex<FPTYPE>) * nstart * ldh, hipMemcpyDeviceToDevice));
        xheevd_wrapper(HIPBLAS_FILL_MODE_LOWER, nstart, V, ldh, W);
    }
};

template struct dngvd_op<float, psi::DEVICE_GPU>;
template struct dnevx_op<float, psi::DEVICE_GPU>;
template struct dngvd_op<double, psi::DEVICE_GPU>;
template struct dnevx_op<double, psi::DEVICE_GPU>;

} // namespace hsolver