#include "hip/hip_runtime.h"
#include "module_hamilt_pw/hamilt_pwdft/kernels/veff_op.h"

#include <complex>
#include <thrust/complex.h>

#include <hip/hip_runtime.h>

namespace hamilt{

#define THREADS_PER_BLOCK 256

template <typename FPTYPE>
__global__ void veff_pw(
    const int size,
    thrust::complex<FPTYPE>* out,
    const FPTYPE* in)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) {return;}
    out[idx] *= in[idx];
}

template <typename FPTYPE>
__global__ void veff_pw(
    const int size,
    thrust::complex<FPTYPE>* out,
    thrust::complex<FPTYPE>* out1,
    const FPTYPE* in)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) {return;}
    thrust::complex<FPTYPE> sup = 
        out[idx] * (in[0 * size + idx] + in[3 * size + idx])
            + out1[idx] * (in[1 * size + idx] - thrust::complex<FPTYPE>(0.0, 1.0) * in[2 * size + idx]);
    thrust::complex<FPTYPE> sdown = 
        out1[idx] * (in[0 * size + idx] - in[3 * size + idx])
            + out[idx] * (in[1 * size + idx] + thrust::complex<FPTYPE>(0.0, 1.0) * in[2 * size + idx]);
    out[idx] = sup;
    out1[idx] = sdown;
}

template <typename FPTYPE>
void veff_pw_op<FPTYPE, psi::DEVICE_GPU>::operator() (
    const psi::DEVICE_GPU* dev,
    const int& size,
    std::complex<FPTYPE>* out,
    const FPTYPE* in)
{
    const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipLaunchKernelGGL(HIP_KERNEL_NAME(veff_pw<FPTYPE>), dim3(block), dim3(THREADS_PER_BLOCK), 0, 0, 
        size, // control params
        reinterpret_cast<thrust::complex<FPTYPE>*>(out), // array of data
        in); // array of data
    // cpu part:
    // for (int ir = 0; ir < size; ++ir)
    // {
    //     out[ir] *= in[ir];
    // }
}

template <typename FPTYPE>
void veff_pw_op<FPTYPE, psi::DEVICE_GPU>::operator() (
    const psi::DEVICE_GPU* dev,
    const int& size,
    std::complex<FPTYPE>* out,
    std::complex<FPTYPE>* out1,
    const FPTYPE** in)
{
    const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipLaunchKernelGGL(HIP_KERNEL_NAME(veff_pw<FPTYPE>), dim3(block), dim3(THREADS_PER_BLOCK), 0, 0, 
        size, // control params
        reinterpret_cast<thrust::complex<FPTYPE>*>(out), // array of data
        reinterpret_cast<thrust::complex<FPTYPE>*>(out1), // array of data
        in[0]); // array of data
    // cpu part:
    // std::complex<FPTYPE> sup = {0, 0}, sdown = {0, 0};
    // for (int ir = 0; ir < size; ir++) {
    //     sup = out[ir] * (in[0][ir] + in[3][ir])
    //         + out1[ir]
    //                 * (in[1][ir]
    //                 - std::complex<FPTYPE>(0.0, 1.0) * in[2][ir]);
    //     sdown = out1[ir] * (in[0][ir] - in[3][ir])
    //             + out[ir]
    //                 * (in[1][ir]
    //                     + std::complex<FPTYPE>(0.0, 1.0) * in[2][ir]);
    //     out[ir] = sup;
    //     out1[ir] = sdown;
    // }
}

template struct veff_pw_op<float, psi::DEVICE_GPU>;
template struct veff_pw_op<double, psi::DEVICE_GPU>;

}  // namespace hamilt